#include "hip/hip_runtime.h"
/* test.cu */

// Libraries :
# include <stdlib.h>
# include <stdio.h>
# include <string.h>
# include <time.h>
# include <ctime>
# include <math.h>
# include <unistd.h>
# include <iostream>
# include <fstream>
using namespace std;

#define NB_THREADS 512


int number_of_blocks(int n)
{
  int res;
  res = n/NB_THREADS;
  if ( n % NB_THREADS != 0)
    res++;
  return res;
}


__global__ void ker(int *T, int n)
{
  int k = blockIdx.x * blockDim.x + threadIdx.x;

  if (k < n)
    T[k] = k;
}


int main()
{
  int n = 512;
  int e = 9;
  int i, nb_blocks;
  int *Td, *Th;

  for (i=0; i<100; i++)
  {
  printf("n = %d\ne = %d\n", n, e);
  Th = (int*) calloc(n, sizeof(int));
  printf("      --> Th calloc done\n");
  hipMalloc( (void **) &Td, n * sizeof(int) );
  hipDeviceSynchronize();
  printf("      --> Td hipMalloc done\n");
  hipMemcpy( Td, Th, n*sizeof(int), hipMemcpyHostToDevice );
  hipDeviceSynchronize();
  printf("      --> hipMemcpy(Td, Th) done\n");
  nb_blocks = number_of_blocks(n);
  ker<<<nb_blocks, NB_THREADS>>>(Td, n);
  printf("      --> ker(Td) done\n");
  hipMemcpy( Th, Td, n*sizeof(int), hipMemcpyDeviceToHost );
  hipDeviceSynchronize();
  printf("      --> hipMemcpy(Th, Td) done\n");
  free(Th);
  hipFree(Td);
  n *= 2;
  e++;
  }

}
