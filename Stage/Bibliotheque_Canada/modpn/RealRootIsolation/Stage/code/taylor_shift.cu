#include "hip/hip_runtime.h"
#include "taylor_shift_conf.h"
#include "inlines.h"
#include "taylor_shift_cpu.h"
#include "taylor_shift_kernel.h"
#include "taylor_shift.h"
#include "taylor_shift_fft.h"
#include "list_pointwise_mul.h"
#include "list_stockham.h"

/* Important to notice :

  n  : number of coefficients of the polynomial considered
 n-1 : degree of the polynomial considered
  p  : prime number, it must be greater than n
*/


// Taylor_shift procedure
void taylor_shift_GPU(sfixn n, sfixn e, char *file, sfixn p, double pinv)
{

  // declaration of variables
  sfixn i, nb_blocks, local_n;
  sfixn *Factorial_device;
  sfixn *Polynomial, *Polynomial_device;
  sfixn *Monomial_shift_device;
  sfixn *temp;
  sfixn *Mgpu;
  sfixn *Polynomial_shift_device[2];
  float cpu_time, gpu_time, outerTime;
  hipEvent_t start, stop;     /* Initial and final time */

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);

  // beginning parameters
  local_n = 2;
  stock_file_in_array(file, n, Polynomial);


  // display parameters
//  printf("\n~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n\
                    TAYLOR_SHIFT ON GPU\n\n\
~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~~\n\n");

//  printf("\nPARAMETERS :\n------------\n\n");
//  printf("  * n = %d\n", n);
//  printf("  * e = %d\n", e);
//  printf("  * p = %d\n", p);
//  printf("  * local_n = %d\n", local_n);
//  printf("  * pinv = %0.20lf\n", pinv);

  // TIME
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&cpu_time, start, stop);
  hipEventDestroy(stop);
  cpu_time /= 1000.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  // Create the array Factorial
  hipMalloc( (void **) &Factorial_device, (n+1) * sizeof(sfixn) );
  nb_blocks = number_of_blocks(n+1);
  hipDeviceSynchronize();
  identity_GPU<<<nb_blocks, NB_THREADS>>>(Factorial_device, n+1);
  hipDeviceSynchronize();



    // display Factorial_device
//    temp = (sfixn*) calloc(n, sizeof(sfixn));
//    hipMemcpy( temp, Factorial_device, n*sizeof(sfixn), hipMemcpyDeviceToHost );
//    printf("\nFactorial_device AFTER IDENTITY_GPU :\n");
//    display_array(temp, n);
//    free(temp);



//  printf("\n      --> identity_GPU done\n");
  nb_blocks = number_of_blocks(n/2);
//  create_factorial_GPU<<<nb_blocks, NB_THREADS>>>(Factorial_device + 1, n, e, p, pinv);
  create_factorial_step0_GPU<<<nb_blocks, NB_THREADS>>>(Factorial_device + 1, n, e, p, pinv);
  hipDeviceSynchronize();
  sfixn L = 1;
  for (i=1; i<e; i++)
  {
    L *= 2;
    create_factorial_stepi_GPU<<<nb_blocks, NB_THREADS>>>(Factorial_device + 1, n, e, p, pinv, L);
    hipDeviceSynchronize();
  }
//  printf("\n      --> create_factorial_GPU done\n");


  // Create the array of the (x+1)^i
  hipMalloc( (void **) &Monomial_shift_device , n * sizeof(sfixn) ); // n+1
  hipDeviceSynchronize();
  nb_blocks = number_of_blocks(n);
  develop_xshift_GPU<<<nb_blocks, NB_THREADS>>>(Monomial_shift_device, n, Factorial_device, p, pinv);
  hipDeviceSynchronize();
  hipFree(Factorial_device);
//  printf("\n      --> develop_xshift_GPU done\n");
//  hipFree(Factorial_device);

    // display Factorial_device
//    temp = (sfixn*) calloc(n, sizeof(sfixn));
//    hipMemcpy( temp, Factorial_device, n*sizeof(sfixn), hipMemcpyDeviceToHost );
//    printf("\nFactorial_device AFTER DEVELOP_XSHIFT_GPU :\n");
//    display_array(temp, n);
//    free(temp);

    // display Monomial_shift_device
//    temp = (sfixn*) calloc(n, sizeof(sfixn));
//    hipMemcpy( temp, Monomial_shift_device, n*sizeof(sfixn), hipMemcpyDeviceToHost );
//    printf("\nMonomial_shift_device :\n");
//    display_array(temp, n);
//    free(temp);



  /* ************************************************************

                     1st step : initialization 

     ************************************************************ */

//  printf("\n\nStep 1 :\n-------- \n");
  hipMalloc( (void **) &Polynomial_device, n * sizeof(sfixn) );
  hipMemcpy( Polynomial_device, Polynomial, n*sizeof(sfixn), hipMemcpyHostToDevice );
  free(Polynomial);
  hipMalloc( (void **) &Polynomial_shift_device[0], n * sizeof(sfixn) );
  hipDeviceSynchronize();

  // initialize polynomial_shift
  nb_blocks = number_of_blocks(n/2);
  init_polynomial_shift_GPU<<<nb_blocks, NB_THREADS>>>(Polynomial_device, Polynomial_shift_device[0], n, p);
  hipDeviceSynchronize();
//  printf("\n      --> init_polynomial_shift_GPU done\n");



  /* ************************************************************

                          next steps (i<10)

     ************************************************************ */

  sfixn polyOnLayerCurrent = n/2;
  sfixn mulInThreadBlock;

  hipMalloc((void **)&Mgpu, n * sizeof(sfixn));

  sfixn I = 9;
  if (e < 9)
    I = e;

  hipMalloc( (void **) &Polynomial_shift_device[1], n * sizeof(sfixn) );
  hipDeviceSynchronize();

  // LOOP
  for (i=1; i<I; i++)
  {
//    printf("\n\nStep %d :\n-------- \n\n", i+1);
//    printf("  * local_n = %d\n", local_n);
//    printf("  * B = %d\n", 2 * local_n);
//    printf("  * polyOnLayerCurrent = %d\n", polyOnLayerCurrent);


    // transfer the polynomials which will be computed
    nb_blocks = number_of_blocks(n);
    transfert_array_GPU<<<nb_blocks, NB_THREADS>>>(Mgpu, Polynomial_shift_device[(i+1)%2], Monomial_shift_device, n, local_n, p, pinv);
    hipDeviceSynchronize();
//    printf("\n      --> transfert_array_GPU on Mgpu done\n");


    // Compute the product of the polynomials in Mgpu ('P2 * Bin' with Bin the array of binomials) and store them in Polynomial_shift_device[i%2] shifted at the right for the multiplication by x so do [( (x+1)^i - 1 ) / x ] * P2(x+1), then multiply it by x so we have [(x+1)^i - 1] * P2(x+1)
    mulInThreadBlock = (sfixn) floor((double) NB_THREADS / (double) (2*local_n));
    nb_blocks = (sfixn) ceil(((double) polyOnLayerCurrent / (double) mulInThreadBlock) * 0.5);
    listPlainMulGpu_and_right_shift_GPU<<<nb_blocks, NB_THREADS>>>(Mgpu, Polynomial_shift_device[i%2], local_n, polyOnLayerCurrent, 2*local_n, mulInThreadBlock, p, pinv);
    hipDeviceSynchronize();
//    printf("\n      --> listPlainMulGpu_and_right_shift_GPU done\n");


    // add [(x+1)^i - 1] * P2(x+1) with P2(x+1) then we get (x+1)^i * P2(x+1) then do P1(x+1) + (x+1)^i*P2(x+1)
    nb_blocks = number_of_blocks(n/2);
    semi_add_GPU<<<nb_blocks, NB_THREADS>>>(Polynomial_shift_device[i%2], Mgpu, Polynomial_shift_device[(i+1)%2], n, local_n, p);
    hipDeviceSynchronize();
//    printf("\n      --> semi_add_GPU done\n");


    // for the next step
    polyOnLayerCurrent /= 2;
    local_n *= 2;
  }



  /* ************************************************************

                       next steps : FFT (i >= 10)

     ************************************************************ */


  sfixn J = e;
  if (e < 9)
    J = 9;
  sfixn w;
  sfixn *fft_device;
  hipMalloc( (void **) &fft_device, 2 * n * sizeof(sfixn) );
  hipDeviceSynchronize();

  // LOOP
  for (i=9; i<J; i++)
  {
//    printf("\n\nStep %d :\n--------- \n\n", i+1);
//    printf("  * local_n = %d\n", local_n);
//    printf("  * B = %d\n", 2 * local_n);
//    printf("  * polyOnLayerCurrent = %d\n", polyOnLayerCurrent);


    // transfer the polynomials which will be FFTed and Mgpu
    nb_blocks = number_of_blocks(n);
    transfert_array_GPU<<<nb_blocks, NB_THREADS>>>(Mgpu, Polynomial_shift_device[(i+1)%2], Monomial_shift_device, n, local_n, p, pinv);
    hipDeviceSynchronize();
    nb_blocks = number_of_blocks(2*n);
    transfert_array_fft_GPU<<<nb_blocks, NB_THREADS>>>(fft_device, Mgpu, n, local_n);
    hipDeviceSynchronize();
//    printf("\n      --> transfert_array_fft_GPU done\n");


    // Convert the polynomials in the FFT world
    w = primitive_root(i+1, p);
    list_stockham_dev(fft_device, polyOnLayerCurrent, i+1, w, p);
    hipDeviceSynchronize();
//    printf("\n      --> list_stockham_dev done\n");


    // same operation than for ListPlainMul but in the FFT world
    nb_blocks = number_of_blocks(2*n);
    list_pointwise_mul<<<nb_blocks, NB_THREADS>>>(fft_device, 2*local_n, p, pinv, 2*n);
    hipDeviceSynchronize();
//    printf("\n      --> list_pointwise_mul done\n");


    // return to the real world
    w = inv_mod(w, p);
    list_stockham_dev(fft_device, polyOnLayerCurrent, i+1, w, p);
    hipDeviceSynchronize();
//    printf("\n      --> list_stockham_dev done\n");


    // adjust the real coefficients : we need to multiplicate by the following w to have to correct size
    w = inv_mod(2*local_n, p);
    nb_blocks = number_of_blocks(n);
    mult_adjust_GPU<<<nb_blocks, NB_THREADS>>>(Polynomial_shift_device[i%2], fft_device, n, local_n, w, p, pinv);
    hipDeviceSynchronize();
//    printf("\n      --> mult_adjust_GPU done\n");


    // semi_add
    nb_blocks = number_of_blocks(n/2);
    semi_add_GPU<<<nb_blocks, NB_THREADS>>>(Polynomial_shift_device[i%2], Mgpu, Polynomial_shift_device[(i+1)%2], n, local_n, p);
    hipDeviceSynchronize();
//    printf("\n      --> semi_add_GPU done\n");


    // for the next steps
    polyOnLayerCurrent /= 2;
    local_n *= 2;
  }


  /* ************************************************************

                         end : results

     ************************************************************ */


  // Copy the last array containing the Taylor shift by 1 of the input polynomial
  temp = (sfixn*) malloc(n * sizeof(sfixn));
  hipMemcpy( temp, Polynomial_shift_device[(e-1)%2], n*sizeof(sfixn), hipMemcpyDeviceToHost );
  hipDeviceSynchronize();


  // TIME
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&gpu_time, start, stop);
  hipEventDestroy(stop);
  gpu_time /= 1000.0;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);


  // stockes the array of Newton's coefficients in a file
  char name_file[100];
  sprintf(name_file, "Pol%d.shiftGPU_%d.dat\0", e, p);
  stock_array_in_file(name_file, temp, n);
//  printf("\n      --> Polynomial_shift_device stored in %s done\n", name_file);


  // deallocation of the last arrays
  free(temp);
  hipFree(Monomial_shift_device);
  hipFree(Mgpu);
  hipFree(fft_device);
  for (i=0; i<2; i++)
    hipFree(Polynomial_shift_device[i]);


  // TIME
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&outerTime, start, stop);
  hipEventDestroy(stop);
  outerTime /= 1000.0;
  cpu_time += outerTime;


  // execution time
//  printf("  * cpu_time = %.6f s\n", cpu_time);
//  printf("  * gpu_time = %.6f s\n", gpu_time);
}
